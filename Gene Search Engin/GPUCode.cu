#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include ""
#define MIN(x,y) ((x) < (y) ? (x) : (y))
#define MAX(x,y) ((x) > (y) ? (x) : (y))


__device__ void NW(char *Adata, char *Bdata, int slice, int z, int blen, int *NewH, int Increment, int Max)
{
	//int i = threadIdx.x;

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;

	if (i <= Max)
	{
		int match = 0;
		int mismatch = 1;

		int startIndex;
		if (z <= 0)
		{
			startIndex = slice;
		}
		else
		{
			startIndex = Increment * z + slice;
		}

		int j = startIndex + (i*Increment);

		int row = j / blen;
		int column = j % blen;

		if (row == 0 || column == 0)
		{
			return;
		}

		int score = (Adata[row - 1] == Bdata[column - 1]) ? match : mismatch;
		NewH[column + row* blen] = MIN(NewH[(column - 1) + (row - 1) * blen] + score, MIN(NewH[(column)+(row - 1) * blen] + 1, NewH[(column - 1) + (row)* blen] + 1));
	}
}

__global__ void init_rows_NW(int *NewH, int blen)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;


	int row = i / blen;
	int column = i % blen;

	if (row == 0 && column>0)
	{
		NewH[column + row* blen] = i;
	}

	if (column == 0 && row>0)
	{
		NewH[column + row* blen] = row;
	}
}

__global__ void init_columns_NW(int *NewH, int blen)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;


	int row = i / blen;
	int column = i % blen;

	if (column == 0 && row>0)
	{
		NewH[column + row* blen] = row;
	}
}

__global__ void StartNW(char *Adata, char *Bdata, int blen, int *NewH, int Increment, int Max)
{
	int loop1 = (blen << 1) - 1;
	for (int slice = 0; slice < loop1; ++slice)
	{
		int z = slice < blen ? 0 : slice - blen + 1;//CPU
		int size = (int)ceil((float)((slice - (z << 1)) + 1));
		NW(Adata, Bdata, slice, z, blen, NewH, Increment, Max);
	}
}

void NW_GPU(char *Adata, char *Bdata, int alen, int blen, int *NewH, clock_t &Begain, clock_t &End, clock_t &CreateBegain, clock_t &CreateEnd)
{
	int *dev_H = 0;
	char *dev_a;
	char *dev_b;
	hipSetDevice(0);
	CreateBegain = clock();
	//Create memory allocation in GPU
	hipMalloc((void**)&dev_H, alen * blen * sizeof(int));
	hipMalloc((void**)&dev_a, (alen-1)  * sizeof(char));
	hipMalloc((void**)&dev_b, (alen - 1)  * sizeof(char));
	CreateEnd = clock();

	Begain = clock();
	//Copy all arrays to GPU memory
//	hipMemcpy(dev_H, NewH, alen * blen * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_a, Adata, (alen - 1)  * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, Bdata, (alen - 1)  * sizeof(char), hipMemcpyHostToDevice);
	
	const int NumberOfThreads = 256;
	dim3 ThreadsWarp(32, 8);
	// Here run initionlize in GPU side
	
	
	init_rows_NW << <ThreadsWarp, alen >> >(dev_H, blen);
	
	//Set GPU for parallel working

	int size = (int)ceil((float)blen / (float)NumberOfThreads);
	int Increment = alen - 1;
	//begain time of GPU
	int MemSize = alen * blen;
	StartNW << <ThreadsWarp, MemSize >> >(dev_a, dev_b, alen, dev_H, Increment, size);
	hipMemcpy(NewH, dev_H, alen * blen  * sizeof(int), hipMemcpyDeviceToHost);
	End = clock();								//End time of GPU

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_H);
	hipDeviceReset();
}

void NW_CPU(char *Adata, char *Bdata, int alen, int blen, int *NewH, clock_t &Begain, clock_t &End)
{
	int i, j;
	int inf = 9999;
	//------------------Initializing The Matricies-------------------
	int score;
	int match = 0;
	int mismatch = 1;
	
	NewH[0] = 0;
	Begain = clock();										//begain time of GPU
	
	for (i = 1; i < blen; ++i)
	{
		NewH[i] = i;
	}

	for (j = 1; j < alen; j++)
	{
		NewH[blen * j] = j;
	}

	Begain = clock();										//begain time of CPU
	for (int slice = 0; slice < 2 * alen - 1; ++slice)
	{
		int z = slice < alen ? 0 : slice - alen + 1;
		for (int j = z; j <= slice - z; ++j)
		{
			int row = j;
			int column = (slice - j);

			if (row == 0 || column == 0)
			{
				continue;
			}
			score = (Adata[row - 1] == Bdata[column - 1]) ? 0 : 1;
			NewH[(column)+row * blen] = MIN(NewH[(column - 1) + (row - 1) * blen] + score, MIN(NewH[(column)+(row - 1) * blen] + 1, NewH[(column - 1) + (row)* blen] + 1));

		}
	}

	End = clock();
}

__global__ void init_SM(int *NewH, int MAX)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;

	if (i <= MAX)
	{
		if (i == 0)
		{
			NewH[0] = 0;
		}
		else
		{
			NewH[i] =0;
			NewH[MAX * i] = 0;
		}
	}
}

__device__ void SM(char *Adata, char *Bdata, int slice, int z, int blen, int *NewH, int Increment, int Max)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;

	if (i <= Max)
	{
		int match = 1;
		int mismatch = -1;
		int Gap = -2;

		int startIndex;
		if (z <= 0)
		{
			startIndex = slice;
		}
		else
		{
			startIndex = Increment * z + slice;
		}

		int j = startIndex + (i*Increment);

		int row = j / blen;
		int column = j % blen;

		if (row == 0 || column == 0)
		{
			return;
		}

		int score = (Adata[row - 1] == Bdata[column - 1]) ? match : mismatch;
		NewH[column + row* blen] = MAX(NewH[(column - 1) + (row - 1) * blen] + score, MAX(NewH[(column - 1) + row * blen] + Gap, NewH[column + (row - 1) * blen] + Gap));
		if (NewH[column + row * blen] <= 0)
			{
				NewH[column + row * blen] = 0;
			}
	}
}

__global__ void StartSW(char *Adata, char *Bdata, int blen, int *NewH, int Increment, int Max)
{
	int loop1 = (blen << 1) - 1;
	for (int slice = 0; slice < loop1; ++slice)
	{
		int z = slice < blen ? 0 : slice - blen + 1;//CPU
		int size = (int)ceil((float)((slice - (z << 1)) + 1));
		SM(Adata, Bdata, slice, z, blen, NewH, Increment, Max);
	}
}
void SM_CPU(char *Adata, char *Bdata, int alen, int blen, int *NewH, clock_t &Begain, clock_t &End)
{
	int score;
	int match = 1;
	int mismatch = -1;
	int Gap = -2;
	int i, j;
	//--------Init----------
	for (i = 0; i < alen; ++i)
	{
		NewH[i] = 0;
	}

	for (j = 0; j < blen; ++j)
	{
		NewH[j*blen] = 0;
	}
	//---------------------

	Begain = clock();										//begain time of CPU
	for (int slice = 0; slice < 2 * alen - 1; ++slice)
	{
		//printf("Slice %d: \n", slice);
		int z = slice < alen ? 0 : slice - alen + 1;
		for (int j = z; j <= slice - z; ++j)
		{
			int row = j;
			int column = (slice - j);

			if (row == 0 || column == 0)
			{
				continue;
			}
			score = (Adata[row - 1] == Bdata[column - 1]) ? match : mismatch;

			NewH[column + row * blen] = MAX(NewH[(column - 1) + (row - 1) * blen] + score, MAX(NewH[(column - 1) + row * blen] + Gap, NewH[column + (row - 1) * blen] + Gap));
			if (NewH[column + row * blen] <= 0)
			{
				NewH[column + row * blen] = 0;
			}
		}
	}
	End = clock();

}

void SM_GPU(char *Adata, char *Bdata, int alen, int blen, int *NewH, clock_t &Begain, clock_t &End, clock_t &CreateBegain, clock_t &CreateEnd)
{

	int i, j;
	int inf = 9999;
	//------------------Initializing The Matricies-------------------

	int *dev_H = 0;

	char *dev_a;
	char *dev_b;
	
	hipSetDevice(0);
	CreateBegain = clock();
	//Create memory allocation in GPU
	hipMalloc((void**)&dev_H, alen * blen * sizeof(int));
	hipMalloc((void**)&dev_a, alen  * sizeof(char));
	hipMalloc((void**)&dev_b, alen  * sizeof(char));
	CreateEnd =  clock();

	//Copy all arrays to GPU memory
	Begain = clock();										//begain time of GPU
	hipMemcpy(dev_a, Adata, alen * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, Bdata, blen * sizeof(char), hipMemcpyHostToDevice);

	//hipMemcpy(dev_H, NewH, alen * blen * sizeof(int), hipMemcpyHostToDevice);
	const int NumberOfThreads = 256;

	//Set GPU for parallel working
	dim3 ThreadsWarp(32, 8);
	int size = (int)ceil((float)blen / (float)NumberOfThreads);
	int Increment = alen - 1;
	int MemSize = alen * blen;
	//int MEM1 = 
	init_SM << < NumberOfThreads, size >> >(dev_H, blen);
	StartSW << <ThreadsWarp, MemSize >> >(dev_a, dev_b, alen, dev_H, Increment, size);
	//for (int slice = 0; slice < 2 * alen - 1; ++slice)
	//{
	//	int z = slice < alen ? 0 : slice - alen + 1;//CPU
	//	size = (int)ceil((float)((slice - 2 * z) + 1));

	//	SM << <ThreadsWarp, MemSize >> >(dev_a, dev_b, slice, z, alen, dev_H, Increment, size);
	//}

	hipMemcpy(NewH, dev_H, alen * blen  * sizeof(int), hipMemcpyDeviceToHost);
	End = clock();								//End time of GPU
	hipFree(dev_a);
	hipFree(dev_b);
	
	hipFree(dev_H);
	hipDeviceReset();
}

// ---------------------Test Section---------------------------------
__global__ void Kernel(int *a, int *b, int *c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


void Kernal_Test(int *a, int *b, int* c,int &count)
{
	int *d_a, *d_b,*d_c;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}
	cudaStatus = hipMalloc((void **)&d_a, count *sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}
	cudaStatus = hipMalloc((void **)&d_b, count *sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}
	cudaStatus = hipMalloc((void **)&d_c, count *sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	hipMemcpy(d_a, a, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, count * sizeof(int), hipMemcpyHostToDevice);
	int blocks = 1;
	Kernel << < blocks, count >> >(d_a, d_b, d_c);
	hipMemcpy(c,d_c , count * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceReset();
	count = 10;
}